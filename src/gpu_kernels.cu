#include "hip/hip_runtime.h"
#include "gpu_kernels.cuh"
#include <cstdio>

namespace janus
{

    // CUDA kernels
    __global__ void hello_gpu_kernel(float *data, int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            data[idx] = data[idx] * 2.0f + 1.0f;
        }
    }

    __global__ void update_positions_kernel(float *positions, float *velocities,
                                            float time_step, int num_particles)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < num_particles)
        {
            // Simple Euler integration
            positions[idx] += velocities[idx] * time_step;
        }
    }

    // Host functions for GPU operations
    hipError_t allocate_device_memory(void **device_ptr, size_t size)
    {
        return hipMalloc(device_ptr, size);
    }

    hipError_t free_device_memory(void *device_ptr)
    {
        return hipFree(device_ptr);
    }

    hipError_t copy_to_device(void *device_ptr, const void *host_ptr, size_t size)
    {
        return hipMemcpy(device_ptr, host_ptr, size, hipMemcpyHostToDevice);
    }

    hipError_t copy_from_device(void *host_ptr, const void *device_ptr, size_t size)
    {
        return hipMemcpy(host_ptr, device_ptr, size, hipMemcpyDeviceToHost);
    }

    // GPU utility functions
    int get_optimal_block_size(int data_size)
    {
        int device;
        hipGetDevice(&device);

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);

        // Use a reasonable block size, not exceeding max threads per block
        int block_size = 256;
        if (block_size > prop.maxThreadsPerBlock)
        {
            block_size = prop.maxThreadsPerBlock;
        }
        return block_size;
    }

    int get_optimal_grid_size(int data_size, int block_size)
    {
        return (data_size + block_size - 1) / block_size;
    }

} // namespace janus