#include "hip/hip_runtime.h"
#include "gpu_kernels.cuh"
#include <cstdio>

namespace janus
{

    // CUDA kernels
    __global__ void hello_gpu_kernel(float *data, int size)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < size)
        {
            data[idx] = data[idx] * 2.0f + 1.0f;
        }
    }

    __global__ void update_positions_kernel(double *x, double *y, double *vx, double *vy,
                                            double *ax, double *ay, double time_step, int num_particles)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < num_particles)
        {
            // Velocity Verlet integration for 2D
            // Half kick: v += 0.5*dt*a
            vx[idx] += 0.5 * time_step * ax[idx];
            vy[idx] += 0.5 * time_step * ay[idx];

            // Drift: x += dt*v
            x[idx] += vx[idx] * time_step;
            y[idx] += vy[idx] * time_step;

            // Note: Second half kick and acceleration update would be done here
            // For now, keeping accelerations as-is (zero forces)
            // vx[idx] += 0.5 * time_step * ax[idx];
            // vy[idx] += 0.5 * time_step * ay[idx];
        }
    }

    // Host functions for GPU operations
    hipError_t allocate_device_memory(void **device_ptr, size_t size)
    {
        return hipMalloc(device_ptr, size);
    }

    hipError_t free_device_memory(void *device_ptr)
    {
        return hipFree(device_ptr);
    }

    hipError_t copy_to_device(void *device_ptr, const void *host_ptr, size_t size)
    {
        return hipMemcpy(device_ptr, host_ptr, size, hipMemcpyHostToDevice);
    }

    hipError_t copy_from_device(void *host_ptr, const void *device_ptr, size_t size)
    {
        return hipMemcpy(host_ptr, device_ptr, size, hipMemcpyDeviceToHost);
    }

    // GPU utility functions
    int get_optimal_block_size(int data_size)
    {
        int device;
        hipGetDevice(&device);

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device);

        // Use a reasonable block size, not exceeding max threads per block
        int block_size = 256;
        if (block_size > prop.maxThreadsPerBlock)
        {
            block_size = prop.maxThreadsPerBlock;
        }
        return block_size;
    }

    int get_optimal_grid_size(int data_size, int block_size)
    {
        return (data_size + block_size - 1) / block_size;
    }

    // High-level GPU kernel launcher
    hipError_t launch_update_positions_kernel(double *d_x, double *d_y, double *d_vx, double *d_vy,
                                               double *d_ax, double *d_ay, double time_step, int num_particles)
    {
        // Compute grid and block dimensions
        int block_size = get_optimal_block_size(num_particles);
        int grid_size = get_optimal_grid_size(num_particles, block_size);

        // Launch GPU kernel
        update_positions_kernel<<<grid_size, block_size>>>(d_x, d_y, d_vx, d_vy, d_ax, d_ay, time_step, num_particles);

        // Check for kernel launch errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            return err;
        }

        // Synchronize to ensure kernel completion
        err = hipDeviceSynchronize();
        return err;
    }

} // namespace janus